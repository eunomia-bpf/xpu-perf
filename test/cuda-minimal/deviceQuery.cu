/**
 * Device Query
 * This sample queries the properties of the CUDA devices present in the system
 * From CUDA SDK Samples - Modified for CUDA 12.8 and 13.0 compatibility
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

// Forward declaration
inline int _ConvertSMVer2Cores(int major, int minor);

int main(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);
    printf(" CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
               (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
               driverVersion / 1000, (driverVersion % 100) / 10,
               runtimeVersion / 1000, (runtimeVersion % 100) / 10);
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
               deviceProp.major, deviceProp.minor);

        char msg[256];
        sprintf(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem / 1048576.0f, (unsigned long long)deviceProp.totalGlobalMem);
        printf("%s", msg);

        printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
               deviceProp.multiProcessorCount,
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
               _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
               deviceProp.multiProcessorCount);

        printf("  Memory Bus Width:                              %d-bit\n",
               deviceProp.memoryBusWidth);

        if (deviceProp.l2CacheSize) {
            printf("  L2 Cache Size:                                 %d bytes\n",
                   deviceProp.l2CacheSize);
        }

        printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
               deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
               deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
        printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
               deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
        printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
               deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);

        printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
        printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n", deviceProp.warpSize);
        printf("  Maximum number of threads per multiprocessor:  %d\n",
               deviceProp.maxThreadsPerMultiProcessor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
        printf("  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
        printf("  Concurrent copy and kernel execution:          Yes with %d copy engine(s)\n",
               deviceProp.asyncEngineCount);
        printf("  Run time limit on kernels:                     %s\n",
               deviceProp.computePreemptionSupported ? "Yes" : "No");
        printf("  Integrated GPU sharing Host Memory:            %s\n",
               deviceProp.integrated ? "Yes" : "No");
        printf("  Support host page-locked memory mapping:       %s\n",
               deviceProp.canMapHostMemory ? "Yes" : "No");
        printf("  Alignment requirement for Surfaces:            %s\n",
               deviceProp.surfaceAlignment ? "Yes" : "No");
        printf("  Device has ECC support:                        %s\n",
               deviceProp.ECCEnabled ? "Enabled" : "Disabled");
        printf("  Device supports Unified Addressing (UVA):      %s\n",
               deviceProp.unifiedAddressing ? "Yes" : "No");
        printf("  Supports Cooperative Kernel Launch:            %s\n",
               deviceProp.cooperativeLaunch ? "Yes" : "No");
        printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n",
               deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);
    }

    printf("\ndeviceQuery, CUDA Driver = CUDART, CUDA Driver Version = %d.%d, CUDA Runtime Version = %d.%d, NumDevs = %d\n",
           driverVersion / 1000, (driverVersion % 100) / 10, 
           runtimeVersion / 1000, (runtimeVersion % 100) / 10,
           deviceCount);
    
    printf("Result = PASS\n");

    return 0;
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
    typedef struct {
        int SM;
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] = {
        {0x30, 192},
        {0x32, 192},
        {0x35, 192},
        {0x37, 192},
        {0x50, 128},
        {0x52, 128},
        {0x53, 128},
        {0x60, 64},
        {0x61, 128},
        {0x62, 128},
        {0x70, 64},
        {0x72, 64},
        {0x75, 64},
        {0x80, 64},
        {0x86, 128},
        {0x87, 128},
        {0x89, 128},
        {0x90, 128},
        {-1, -1}
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }

    printf("MapSMtoCores for SM %d.%d is undefined. Default to use 64 Cores/SM\n", major, minor);
    return 64;
}