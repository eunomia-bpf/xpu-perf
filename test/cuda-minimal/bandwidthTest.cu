/**
 * Bandwidth Test
 * This test measures host to device and device to device copy bandwidth
 * for pageable and pinned memory of various sizes.
 * From CUDA SDK Samples
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define MEMCOPY_ITERATIONS 100
#define DEFAULT_SIZE       (32 * (1<<20))
#define DEFAULT_INCREMENT  (1<<22)

#define CACHE_CLEAR_SIZE (1<<24)

const char *sMemoryCopyKind[] = {
    "Host to Device",
    "Device to Host",
    "Device to Device"
};

const char *sMemoryMode[] = {
    "PAGEABLE",
    "PINNED"
};

void printResultsReadable(unsigned int *memSizes, double *bandwidths, 
                          unsigned int count, hipMemcpyKind kind, 
                          int memMode, int iNumDevs)
{
    printf("Bandwidth Test %s Memory Transfers\n", sMemoryMode[memMode]);
    printf("   Transfer Size (Bytes)\tBandwidth(MB/s)\n");
    
    for (unsigned int i = 0; i < count; i++) {
        printf("   %u\t\t\t%s%.1f\n", memSizes[i], 
               (bandwidths[i] < 10000.0) ? "\t" : "",
               bandwidths[i]);
    }
    printf("\n");
}

void testBandwidth(unsigned int memSize, hipMemcpyKind kind, int memMode)
{
    int iNumDevs = 0;
    checkCudaErrors(hipGetDeviceCount(&iNumDevs));
    
    if (iNumDevs == 0) {
        printf("No CUDA-capable devices found.\n");
        exit(EXIT_FAILURE);
    }
    
    unsigned char *h_data = NULL;
    unsigned char *h_cacheClear = NULL;
    unsigned char *d_data = NULL;
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    
    checkCudaErrors(hipMalloc((void **)&d_data, memSize));
    
    if (memMode == 1) {
        checkCudaErrors(hipHostMalloc((void **)&h_data, memSize));
        checkCudaErrors(hipHostMalloc((void **)&h_cacheClear, CACHE_CLEAR_SIZE));
    } else {
        h_data = (unsigned char *)malloc(memSize);
        h_cacheClear = (unsigned char *)malloc(CACHE_CLEAR_SIZE);
        
        if (!h_data || !h_cacheClear) {
            fprintf(stderr, "Failed to allocate host memory\n");
            exit(EXIT_FAILURE);
        }
    }
    
    for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++) {
        h_data[i] = (unsigned char)(i & 0xff);
    }
    
    for (unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(unsigned char); i++) {
        h_cacheClear[i] = (unsigned char)(i & 0xff);
    }
    
    checkCudaErrors(hipEventRecord(start, 0));
    
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
        if (kind == hipMemcpyHostToDevice) {
            checkCudaErrors(hipMemcpy(d_data, h_data, memSize, hipMemcpyHostToDevice));
        } else if (kind == hipMemcpyDeviceToHost) {
            checkCudaErrors(hipMemcpy(h_data, d_data, memSize, hipMemcpyDeviceToHost));
        }
    }
    
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    
    float elapsedTimeInMs = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    
    double elapsedTimeInSec = elapsedTimeInMs / 1000.0;
    double bandwidth = ((double)memSize * (double)MEMCOPY_ITERATIONS) /
                      (elapsedTimeInSec * (double)(1<<20));
    
    unsigned int memSizes[] = {memSize};
    double bandwidths[] = {bandwidth};
    printResultsReadable(memSizes, bandwidths, 1, kind, memMode, iNumDevs);
    
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    
    if (memMode == 1) {
        checkCudaErrors(hipHostFree(h_data));
        checkCudaErrors(hipHostFree(h_cacheClear));
    } else {
        free(h_data);
        free(h_cacheClear);
    }
    
    checkCudaErrors(hipFree(d_data));
}

int main(int argc, char **argv)
{
    printf("[Bandwidth Test] - Starting...\n");
    
    int device = 0;
    checkCudaErrors(hipSetDevice(device));
    
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
    printf("Device %d: %s\n", device, deviceProp.name);
    
    unsigned int memSize = DEFAULT_SIZE;
    
    printf("\nQuick Mode\n");
    testBandwidth(memSize, hipMemcpyHostToDevice, 0);
    testBandwidth(memSize, hipMemcpyDeviceToHost, 0);
    
    printf("\nPinned Memory Transfers\n");
    testBandwidth(memSize, hipMemcpyHostToDevice, 1);
    testBandwidth(memSize, hipMemcpyDeviceToHost, 1);
    
    printf("\nTest passed\n");
    
    return 0;
}