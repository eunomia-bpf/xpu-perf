#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

__global__ void vectorSubtract(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] - b[i];
    }
}

int main() {
    int n = 1000;
    size_t size = n * sizeof(float);

    float *h_a = (float*)malloc(size);
    float *h_b = (float*)malloc(size);
    float *h_c = (float*)malloc(size);

    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Create CUDA graph
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t memcpyNode1, memcpyNode2, memcpyNode3;
    hipGraphNode_t kernelNode1, kernelNode2;
    hipStream_t stream;

    hipStreamCreate(&stream);
    hipGraphCreate(&graph, 0);

    // Add memcpy nodes for input data (H2D)
    hipMemcpy3DParms memcpyParams1 = {0};
    memcpyParams1.srcPtr.ptr = h_a;
    memcpyParams1.dstPtr.ptr = d_a;
    memcpyParams1.extent.width = size;
    memcpyParams1.extent.height = 1;
    memcpyParams1.extent.depth = 1;
    memcpyParams1.kind = hipMemcpyHostToDevice;
    hipGraphAddMemcpyNode(&memcpyNode1, graph, NULL, 0, &memcpyParams1);

    hipMemcpy3DParms memcpyParams2 = {0};
    memcpyParams2.srcPtr.ptr = h_b;
    memcpyParams2.dstPtr.ptr = d_b;
    memcpyParams2.extent.width = size;
    memcpyParams2.extent.height = 1;
    memcpyParams2.extent.depth = 1;
    memcpyParams2.kind = hipMemcpyHostToDevice;
    hipGraphAddMemcpyNode(&memcpyNode2, graph, NULL, 0, &memcpyParams2);

    // Add kernel nodes
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    void* kernelArgs1[] = {&d_a, &d_b, &d_c, &n};
    hipKernelNodeParams kernelParams1 = {0};
    kernelParams1.func = (void*)vectorAdd;
    kernelParams1.gridDim = dim3(numBlocks, 1, 1);
    kernelParams1.blockDim = dim3(blockSize, 1, 1);
    kernelParams1.sharedMemBytes = 0;
    kernelParams1.kernelParams = kernelArgs1;
    kernelParams1.extra = NULL;

    hipGraphNode_t deps1[] = {memcpyNode1, memcpyNode2};
    hipGraphAddKernelNode(&kernelNode1, graph, deps1, 2, &kernelParams1);

    // Add second kernel node (vectorSubtract)
    void* kernelArgs2[] = {&d_a, &d_b, &d_c, &n};
    hipKernelNodeParams kernelParams2 = {0};
    kernelParams2.func = (void*)vectorSubtract;
    kernelParams2.gridDim = dim3(numBlocks, 1, 1);
    kernelParams2.blockDim = dim3(blockSize, 1, 1);
    kernelParams2.sharedMemBytes = 0;
    kernelParams2.kernelParams = kernelArgs2;
    kernelParams2.extra = NULL;

    hipGraphNode_t deps2[] = {kernelNode1};
    hipGraphAddKernelNode(&kernelNode2, graph, deps2, 1, &kernelParams2);

    // Add memcpy node for output data (D2H)
    hipMemcpy3DParms memcpyParams3 = {0};
    memcpyParams3.srcPtr.ptr = d_c;
    memcpyParams3.dstPtr.ptr = h_c;
    memcpyParams3.extent.width = size;
    memcpyParams3.extent.height = 1;
    memcpyParams3.extent.depth = 1;
    memcpyParams3.kind = hipMemcpyDeviceToHost;

    hipGraphNode_t deps3[] = {kernelNode2};
    hipGraphAddMemcpyNode(&memcpyNode3, graph, deps3, 1, &memcpyParams3);

    // Instantiate and launch the graph
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    hipGraphLaunch(graphExec, stream);
    hipStreamSynchronize(stream);

    printf("Verification: h_c[0] = %f (expected -0), h_c[999] = %f (expected -999)\n",
           h_c[0], h_c[999]);

    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
